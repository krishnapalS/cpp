/* Name : Krishna Pal Deora    Admission No. : 18JE0425
 * File:     matrix_addition.cu
 * Program :  Implement matrix addition on a GPU using CUDA
 *
 *
 * Input:    The matrices A and B
 * Output:   Result of matrix addition.  
 *
 */
 
 

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


__global__ void Mat_add(float A[], float B[], float C[], int m, int n) {
   int my_ij = blockDim.x * blockIdx.x + threadIdx.x;

   if (blockIdx.x < m && threadIdx.x < n) 
      C[my_ij] = A[my_ij] + B[my_ij];
}  /* Mat_add */


/*---------------------------------------------------------------------
 * Function:  Read_matrix
 * Purpose:   Read an m x n matrix from stdin
 * In args:   m, n
 * Out arg:   A
 */
void Read_matrix(float A[], int m, int n) {
   int i, j;

   for (i = 0; i < m; i++)
      for (j = 0; j < n; j++)
         scanf("%f", &A[i*n+j]);
}  /* Read_matrix */


/*---------------------------------------------------------------------
 * Function:  Print_matrix
 * Purpose:   Print an m x n matrix to stdout
 * In args:   title, A, m, n
 */
void Print_matrix(char title[], float A[], int m, int n) {
   int i, j;

   printf("%s\n", title);
   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++)
         printf("%.1f ", A[i*n+j]);
      printf("\n");
   }  
}  /* Print_matrix */


/* Host code */
int main(int argc, char* argv[]) {
   int m, n;
   float *h_A, *h_B, *h_C;
   float *d_A, *d_B, *d_C;
   size_t size;

   /* Get size of matrices */
   if (argc != 3) {
      fprintf(stderr, "usage: %s <row count> <col count>\n", argv[0]);
      exit(0);
   }
   m = strtol(argv[1], NULL, 10);
   n = strtol(argv[2], NULL, 10);
   printf("m = %d, n = %d\n", m, n);
   size = m*n*sizeof(float);

   h_A = (float*) malloc(size);
   h_B = (float*) malloc(size);
   h_C = (float*) malloc(size);
   
   printf("Enter the matrices A and B\n");
   Read_matrix(h_A, m, n);
   Read_matrix(h_B, m, n);

   Print_matrix("A =", h_A, m, n);
   Print_matrix("B =", h_B, m, n);

   /* Allocate matrices in device memory */
   hipMalloc(&d_A, size);
   hipMalloc(&d_B, size);
   hipMalloc(&d_C, size);

   /* Copy matrices from host memory to device memory */
   hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

   /* Invoke kernel using m thread blocks, each of    */
   /* which contains n threads                        */
   Mat_add<<<m, n>>>(d_A, d_B, d_C, m, n);


   /* Copy result from device memory to host memory */
   hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

   Print_matrix("The sum is: ", h_C, m, n);

   /* Free device memory */
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   /* Free host memory */
   free(h_A);
   free(h_B);
   free(h_C);

   return 0;
}  /* main */